// GPU implementation of OPQ matrix update, based on NVIDIA cusolver sample code: https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/Xgesvd/cusolver_Xgesvd_example.cu
#include "inc/Quantizer/OPQUpdate.hxx"

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "inc/Quantizer/cusolver_utils.h"
#include <hipblas.h>

int OPQRotationUpdate(float* svd_mat, float* rotation, SPTAG::SizeType dim)
{
  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;

  std::vector<float> U(dim*dim);
  std::vector<float> VT(dim*dim);
  std::vector<float> S(dim);
  std::vector<float> S_exact(dim);

  
  float *d_svd = nullptr;
  float *d_S = nullptr;
  float *d_U = nullptr;
  float *d_VT = nullptr;
  int *d_info = nullptr;
  float *d_work = nullptr;
  float *d_rwork = nullptr;
  float *d_W = nullptr; // W = S*VT

  
  int lwork = 0;
  int info = 0;
  const float h_one = 1;
  const float h_zero = 0;
  const float h_minus_one = -1;

  /* step 1: create cusolver handle, bind a stream */
  CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
  CUBLAS_CHECK(hipblasCreate(&cublasH));

  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
  CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

  
  /* step 2: copy A to device */
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_svd), sizeof(float) * dim * dim));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(float) * S.size()));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(float) * U.size()));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VT), sizeof(float) * VT.size()));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(float) * dim * dim));

  CUDA_CHECK(hipMemcpyAsync(d_svd, svd_mat, sizeof(float) * dim * dim, hipMemcpyHostToDevice, stream));

  /* step 3: query working space of SVD */
  CUSOLVER_CHECK(hipsolverDnDgesvd_bufferSize(cusolverH, dim, dim, &lwork));

  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(float) * lwork));

  
  /* step 4: compute SVD */
  signed char jobu = 'A';  // all dim columns of U
  signed char jobvt = 'A'; // all dim columns of VT

  CUSOLVER_CHECK(hipsolverDnSgesvd(cusolverH, jobu, jobvt, dim, dim, d_svd, dim, d_S, d_U,
								  dim, 
								  d_VT,
								  dim,
								  d_work, lwork, d_rwork, d_info));

  
  CUDA_CHECK(hipMemcpyAsync(U.data(), d_U, sizeof(float) * U.size(), hipMemcpyDeviceToHost,
							 stream));
  CUDA_CHECK(hipMemcpyAsync(VT.data(), d_VT, sizeof(float) * VT.size(),
							 hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(S.data(), d_S, sizeof(float) * S.size(), hipMemcpyDeviceToHost,
							 stream));
  CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));

  // rotation matrix = U * VT
  float *d_rot = nullptr;
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_rot), sizeof(float) * dim * dim));
  
  CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &h_one, d_U, dim, d_VT, dim, &h_zero, d_rot, dim));

  CUDA_CHECK(hipMemcpyAsync(rotation, d_rot, sizeof(float)*dim*dim, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipStreamSynchronize(stream));

  /* free resources */
  CUDA_CHECK(hipFree(d_svd));
  CUDA_CHECK(hipFree(d_S));
  CUDA_CHECK(hipFree(d_U));
  CUDA_CHECK(hipFree(d_VT));
  CUDA_CHECK(hipFree(d_info));
  CUDA_CHECK(hipFree(d_work));
  CUDA_CHECK(hipFree(d_rwork));
  CUDA_CHECK(hipFree(d_W));
  CUDA_CHECK(hipFree(d_rot));

  
  CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
  CUBLAS_CHECK(hipblasDestroy(cublasH));

  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipDeviceReset());

  return EXIT_SUCCESS;
}